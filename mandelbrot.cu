#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// PNG include
#include <png.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 128 // number of threads in each block
#endif

#ifndef IMAGE_SIZE
#define IMAGE_SIZE 4096 // Width and height of output image
#endif

#define DATASET_SIZE (IMAGE_SIZE * IMAGE_SIZE)


typedef png_color* colorp;


uint8_t hEscapeNumber[IMAGE_SIZE * IMAGE_SIZE];
double  hComplexArray[2 * IMAGE_SIZE * IMAGE_SIZE];

void write_palette(colorp palette, uint16_t const palette_size)
{
    uint8_t const Ored   = 244;
    uint8_t const Ogreen = 172;
    uint8_t const Oblue  = 123;
    uint8_t const Bred   = 14;
    uint8_t const Bgreen = 59;
    uint8_t const Bblue  = 92;
    for(size_t i = 0; i < palette_size; i++)
    {
        double scale = (double)i / (double)palette_size;
        double Cred   = 0;
        double Cgreen = 0;
        double Cblue  = 0;
        if(i >= 64 && i < 192) // Orange to blue
        {
            scale  = (double)(i - 64) / (double)128;
            Cred   = (1.0 - scale) * Ored   + scale * Bred;
            Cgreen = (1.0 - scale) * Ogreen + scale * Bgreen;
            Cblue  = (1.0 - scale) * Oblue  + scale * Bblue;
        } else // Blue to orange
        {
            if (i < 64)
            {
                scale = (double)(i + 64) / (double)128;
            } else
            {
                scale = (double)(i - 192) / (double)128;
            }
            Cred   = (1.0 - scale) * Bred   + scale * Ored;
            Cgreen = (1.0 - scale) * Bgreen + scale * Ogreen;
            Cblue  = (1.0 - scale) * Bblue  + scale * Oblue;
        }
        colorp col = &palette[i];
        col->red   = (uint8_t) Cred;
        col->green = (uint8_t) Cgreen;
        col->blue  = (uint8_t) Cblue;
    }
}

int mandelbrot_to_png(char const * const fileName, uint8_t const * const escapeArray)
{
    FILE *fp = fopen(fileName, "wb");
    if (fp == NULL)
    {
        return (1);
    }
    png_structp png_ptr = png_create_write_struct (PNG_LIBPNG_VER_STRING, NULL,
                                                   NULL, NULL);
    if (png_ptr == NULL)
    {
        fclose(fp);
        return (1);
    }
    png_infop info_ptr = png_create_info_struct(png_ptr);
    if (info_ptr == NULL)
    {
        fclose(fp);
        png_destroy_write_struct(&png_ptr, NULL);
        return (1);
    }
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, IMAGE_SIZE, IMAGE_SIZE,
                 8, PNG_COLOR_TYPE_PALETTE, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT,  PNG_FILTER_TYPE_DEFAULT);

    uint16_t const palette_size = 256;
    assert(palette_size <= PNG_MAX_PALETTE_LENGTH);

    png_colorp palette = (png_colorp)png_malloc(png_ptr, palette_size * sizeof (png_color));
    write_palette(palette, palette_size);
    png_set_PLTE(png_ptr, info_ptr, palette, palette_size);
    png_write_info(png_ptr, info_ptr);
    for (size_t i = 0; i < IMAGE_SIZE; i++)
    {
        png_write_row(png_ptr, (png_const_bytep)&escapeArray[i*IMAGE_SIZE]);
    }
    png_write_flush(png_ptr);
    png_write_end(png_ptr, NULL);
    png_free(png_ptr, palette);
    palette=NULL;
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
    return (0);
}


__global__ void generate_complex_array (double const centerReal, double const centerImaginary,
                                        double const scale,      double * const complexArray)
{
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    if( gid < DATASET_SIZE )
    {
        int row               = gid / IMAGE_SIZE;
        int column            = gid % IMAGE_SIZE;
        complexArray[2*gid]   = scale * (column - IMAGE_SIZE/2 ) + centerReal;      // Real
        complexArray[2*gid+1] = scale * (IMAGE_SIZE/2 - row ) + centerImaginary; // Imaginary
    }
}

__global__ void brot_escape( uint8_t const        threshold,    uint32_t const  limit,
                             double const * const complexArray, uint8_t * const escapeNumber)
{
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    if( gid < DATASET_SIZE )
    {
        double const x0 = complexArray[2 * gid];
        double const y0 = complexArray[2 * gid + 1];
        double x2 = 0;
        double y2 = 0;
        double x = 0;
        double y = 0;
        uint16_t iteration = 0;
        while (x2 + y2 < threshold && iteration < limit)
        {
            y = 2 * x * y + y0;
            x = x2 - y2 + x0;
            x2 = x * x;
            y2 = y * y;
            iteration++;
        }
        double palette_index = pow(((double)iteration / (double)limit * 255.0), 1.5); // 2^8 - 1 for 8 bit palette
        escapeNumber[gid] = (uint8_t) palette_index % 255;
    }
}

int
main(int argc, char *argv[])
{
    // int dev = findCudaDevice(argc, (const char **)argv);
    int returnValue = 0;
    if(argc >= 7)
    {
        double      const centerReal      = strtod(argv[1], NULL);
        double      const centerImaginary = strtod(argv[2], NULL);
        uint8_t     const threshold       = atoi(argv[3]);
        uint32_t    const limit           = atoi(argv[4]);
        double      const scale           = strtod(argv[5], NULL);
        char const* const filename        = argv[6];

        // printf("Center at %lf + %lfi, threshold: %d, limit: %d, scale: %lf\n", centerReal, centerImaginary, threshold, limit, scale);

        // allocate device memory:
        double  *dComplexArray;
        uint8_t *dEscapeNumber;

        hipError_t status;
        status = hipMalloc( (void **)(&dComplexArray), sizeof(hComplexArray) );
        checkCudaErrors(status);

        status = hipMalloc( (void **)(&dEscapeNumber), sizeof(hEscapeNumber) );
        checkCudaErrors(status);


        // copy host memory to the device:
        status = hipMemcpy( dEscapeNumber, hEscapeNumber, sizeof(hEscapeNumber), hipMemcpyHostToDevice );
        checkCudaErrors(status);

        // setup the execution parameters:
        dim3 grid(DATASET_SIZE / THREADS_PER_BLOCK, 1, 1 );
        dim3 threads(THREADS_PER_BLOCK, 1, 1 );

        // execute the kernel:
        generate_complex_array<<< grid, threads >>>(centerReal, centerImaginary,
                                                    scale,      dComplexArray);
        status = hipMemcpy( hComplexArray, dComplexArray, sizeof(hComplexArray), hipMemcpyDeviceToHost );
        checkCudaErrors(status);

        brot_escape<<< grid, threads >>>(threshold,  limit,
                                         dComplexArray, dEscapeNumber);

        // copy result from the device to the host:
        status = hipMemcpy( hEscapeNumber, dEscapeNumber, sizeof(hEscapeNumber), hipMemcpyDeviceToHost );
        checkCudaErrors(status);

        status = hipDeviceSynchronize();
        checkCudaErrors(status);

        // clean up:
        status = hipFree( dComplexArray );
        checkCudaErrors(status);

        status = hipFree( dEscapeNumber );
        checkCudaErrors(status);

        return mandelbrot_to_png(filename, hEscapeNumber);
    } else
    {
        fprintf(stderr, "Arguments: center real, center imaginary, threshold, limit, scale, filename\n");
        returnValue = 1;
    }
    return returnValue;
}
